#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/AccumulateType.h"
#include "ATen/cuda/HIPContext.h"
#include <THC/THCDeviceUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include "type_shim.h"

namespace {
// This is the un-specialized struct.  Note that we prevent instantiation of this
// struct by putting an undefined symbol in the function body so it won't compile.
//  template <typename T>
//  struct SharedMemory
//  {
//      // Ensure that we won't compile any un-specialized types
//      __device__ T *getPointer()
//      {
//          extern __device__ void error(void);
//          error();
//          return NULL;
//      }
//  };
// https://github.com/NVIDIA/apex/issues/246
template <typename T>
struct SharedMemory;

template <>
struct SharedMemory <float>
{
    __device__ float *getPointer()
    {
        extern __shared__ float s_float[];
        return s_float;
    }
};

template <>
struct SharedMemory <double>
{
    __device__ double *getPointer()
    {
        extern __shared__ double s_double[];
        return s_double;
    }
};
}

template<typename T, typename U> __device__
void cuLoadWriteStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    U* warp_buf1,
    U* warp_buf2,
    const T* input,
    const T* dout,
    const int i1_end,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar
    )
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    U curr_mean = mean[i1];
    U curr_invvar = invvar[i1];
    for (int k = 0;  k < blockDim.y;  ++k) {
      const int i2 = i2_off + k;
      const int load_idx = i1*n2+i2;
      const int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<n2) {
        U curr_input = static_cast<U>(input[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        warp_buf1[write_idx] = curr_dout;
        warp_buf2[write_idx] = curr_dout * (curr_input - curr_mean) * curr_invvar;
      } else {
        warp_buf1[write_idx] = U(0);
        warp_buf2[write_idx] = U(0);
      }
    }
  } else {
    for (int k = 0;  k < blockDim.y;  ++k) {
      const int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      warp_buf1[write_idx] = U(0);
      warp_buf2[write_idx] = U(0);
    }
  }
}

template<typename T, typename U> __device__
void cuLoadAddStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    U* warp_buf1,
    U* warp_buf2,
    const T* input,
    const T* dout,
    const int i1_end,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar
    )
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    U curr_mean = mean[i1];
    U curr_invvar = invvar[i1];
    for (int k = 0;  k < blockDim.y;  ++k) {
      const int i2 = i2_off + k;
      const int load_idx = i1*n2+i2;
      const int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<n2) {
        U curr_input = static_cast<U>(input[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        warp_buf1[write_idx] += curr_dout;
        warp_buf2[write_idx] += curr_dout * (curr_input - curr_mean) * curr_invvar;
      }
    }
  }
}

template<typename T, typename U> __global__
void cuComputePartGradGammaBeta(
    const T* __restrict__ dout,
    const T* __restrict__ input,
    const int n1,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    U epsilon,
    U* part_grad_gamma,
    U* part_grad_beta)
{
    const int numsegs_n1 = (n1+blockDim.y*blockDim.y-1) / (blockDim.y*blockDim.y);
    const int segs_per_block = (numsegs_n1 + gridDim.y - 1) / gridDim.y;
    const int i1_beg = blockIdx.y * segs_per_block * blockDim.y*blockDim.y;
    const int i1_beg_plus_one = (blockIdx.y+1) * segs_per_block * blockDim.y*blockDim.y;
    const int i1_end = i1_beg_plus_one < n1 ? i1_beg_plus_one : n1;
    const int row_stride = blockDim.x+1;
    const int thr_load_col_off = (threadIdx.x*blockDim.y)&(blockDim.x-1);
    const int thr_load_row_off = (threadIdx.x*blockDim.y)/blockDim.x + threadIdx.y*blockDim.y;
    const int i2_off = blockIdx.x * blockDim.x + thr_load_col_off;
    SharedMemory<U> shared;
    U* buf = shared.getPointer(); // buf has at least blockDim.x * blockDim.y * blockDim.y + (blockDim.y - 1)*(blockDim.x/blockDim.y) elements
    U* warp_buf1 = (U*)buf;
    U* warp_buf2 = warp_buf1 + blockDim.y * blockDim.y * row_stride;
    // compute partial sums from strided inputs
    // do this to increase number of loads in flight
    cuLoadWriteStridedInputs(i1_beg,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf1,warp_buf2,input,dout,i1_end,n2,mean,invvar);
    for (int i1_block = i1_beg+blockDim.y*blockDim.y;  i1_block < i1_end;  i1_block+=blockDim.y*blockDim.y) {
      cuLoadAddStridedInputs(i1_block,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf1,warp_buf2,input,dout,i1_end,n2,mean,invvar);
    }
    __syncthreads();
    // inter-warp reductions
    // sum within each warp
    U acc1 = U(0);
    U acc2 = U(0);
    for (int k = 0;  k < blockDim.y;  ++k) {
      const int row1 = threadIdx.y + k*blockDim.y;
      const int idx1 = row1*row_stride + threadIdx.x;
      acc1 += warp_buf1[idx1];
      acc2 += warp_buf2[idx1];
    }
    warp_buf1[threadIdx.y*row_stride+threadIdx.x] = acc1;
    warp_buf2[threadIdx.y*row_stride+threadIdx.x] = acc2;
    __syncthreads();
    // sum all warps
    for (int offset = blockDim.y/2;  offset > 1;  offset /= 2) {
      if (threadIdx.y < offset) {
        const int row1 = threadIdx.y;
        const int row2 = threadIdx.y + offset;
        const int idx1 = row1*row_stride + threadIdx.x;
        const int idx2 = row2*row_stride + threadIdx.x;
        warp_buf1[idx1] += warp_buf1[idx2];
        warp_buf2[idx1] += warp_buf2[idx2];
      }
      __syncthreads();
    }
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.y == 0 && i2 < n2) {
      const int row1 = threadIdx.y;
      const int row2 = threadIdx.y + 1;
      const int idx1 = row1*row_stride + threadIdx.x;
      const int idx2 = row2*row_stride + threadIdx.x;
      part_grad_beta[blockIdx.y*n2+i2] = warp_buf1[idx1] + warp_buf1[idx2];
      part_grad_gamma[blockIdx.y*n2+i2] = warp_buf2[idx1] + warp_buf2[idx2];
    }
}

template<typename T, typename U> __global__
void cuComputeGradGammaBeta(
    const U* part_grad_gamma,
    const U* part_grad_beta,
    const int part_size,
    const int n1,
    const int n2,
    T* grad_gamma,
    T* grad_beta)
{
    // sum partial gradients for gamma and beta
    SharedMemory<U> shared;
    U* buf = shared.getPointer(); 
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (i2 < n2) {
      // each warp does sequential reductions until reduced part_size is num_warps
      int num_warp_reductions = part_size / blockDim.y;
      U sum_gamma = U(0);
      U sum_beta = U(0);
      const U* part_grad_gamma_ptr = part_grad_gamma + threadIdx.y * num_warp_reductions * n2 + i2;
      const U* part_grad_beta_ptr = part_grad_beta + threadIdx.y * num_warp_reductions * n2 + i2;
      for (int warp_offset = 0;  warp_offset < num_warp_reductions;  ++warp_offset) {
        sum_gamma += part_grad_gamma_ptr[warp_offset*n2];
        sum_beta += part_grad_beta_ptr[warp_offset*n2];
      }
      // inter-warp reductions
      const int nbsize3 = blockDim.x * blockDim.y / 2;
      for (int offset = blockDim.y/2;  offset >= 1;  offset /= 2) {
        // top half write to shared memory
        if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int write_idx = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          buf[write_idx] = sum_gamma;
          buf[write_idx+nbsize3] = sum_beta;
        }
        __syncthreads();
        // bottom half sums
        if (threadIdx.y < offset) {
          const int read_idx = threadIdx.y * blockDim.x + threadIdx.x;
          sum_gamma += buf[read_idx];
          sum_beta += buf[read_idx+nbsize3];
        }
        __syncthreads();
      }
      // write out fully summed gradients
      if (threadIdx.y == 0) {
        grad_gamma[i2] = sum_gamma;
        grad_beta[i2] = sum_beta;
      }
    }
}

template<typename T, typename U> 
void HostLayerNormGradient(
    const T* dout,
    const U* mean,
    const U* invvar,
    at::Tensor* input,
    int n1,
    int n2,
    const T* gamma,
    const T* beta,
    double epsilon,
    T* grad_gamma,
    T* grad_beta
    )
{
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    if (gamma != NULL && beta != NULL) {
      // compute grad_gamma(j) and grad_beta(j)
      const int part_size = 16;
      const dim3 threads2(32,4,1);
      const dim3 blocks2((n2+threads2.x-1)/threads2.x,part_size,1);
      const int nshared2_a = 2 * sizeof(U) * threads2.y * threads2.y * (threads2.x + 1);
      const int nshared2_b = threads2.x * threads2.y * sizeof(U);
      const int nshared2 = nshared2_a > nshared2_b ? nshared2_a : nshared2_b;
      at::Tensor part_grad_gamma = at::empty({part_size,n2}, input->options().dtype((input->scalar_type()==at::ScalarType::Half || input->scalar_type()==at::ScalarType::BFloat16) ? at::ScalarType::Float : input->scalar_type()));
      at::Tensor part_grad_beta = at::empty_like(part_grad_gamma);
      cuComputePartGradGammaBeta<<<blocks2, threads2, nshared2, stream>>>(
              dout,
              input->data_ptr<T>(),
              n1,n2,
              mean,
              invvar,
              U(epsilon),
              part_grad_gamma.data_ptr<U>(),
              part_grad_beta.data_ptr<U>());

      const dim3 threads3(32,8,1);
      const dim3 blocks3((n2+threads2.x-1)/threads2.x,1,1);
      const int nshared3 = threads3.x * threads3.y * sizeof(U);
      cuComputeGradGammaBeta<<<blocks3, threads3, nshared3, stream>>>(
              part_grad_gamma.data_ptr<U>(),
              part_grad_beta.data_ptr<U>(),
              part_size,
              n1,n2,
              grad_gamma,
              grad_beta);
    }
}

void cuda_layer_norm_gradient(
    at::Tensor* dout,
    at::Tensor* mean,
    at::Tensor* invvar,
    at::Tensor* input,
    int n1,
    int n2,
    #ifdef VERSION_GE_1_1
    at::IntArrayRef normalized_shape,
    #else
    at::IntList normalized_shape,
    #endif
    at::Tensor* gamma,
    at::Tensor* beta,
    double epsilon,
    at::Tensor* grad_gamma,
    at::Tensor* grad_beta)
{
    using namespace at;
    DISPATCH_DOUBLE_FLOAT_AND_HALF_AND_BF16(input->scalar_type(), 0, "cuComputeGradInput",
        using accscalar_t = at::acc_type<scalar_t_0, true>;
        HostLayerNormGradient(
        dout->data_ptr<scalar_t_0>(),
        mean->data_ptr<accscalar_t>(),
        invvar->data_ptr<accscalar_t>(),
        input,
        n1,n2,
            // TMJ pass NULL argument for gamma, beta, grad_gamma and grad_beta
            // if gamma Tensor is NULL on input.
        gamma->data_ptr<scalar_t_0>(),
        beta->data_ptr<scalar_t_0>(),
        epsilon,
        grad_gamma->data_ptr<scalar_t_0>(),
        grad_beta->data_ptr<scalar_t_0>());
      )
}